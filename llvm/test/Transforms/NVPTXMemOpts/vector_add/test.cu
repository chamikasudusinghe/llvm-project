
#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA kernel for vector addition
__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements) {
        C[i] = A[i] + B[i];
    }
}

// Cuda kernel for vector addition with memory coalescing
__global__ void vectorAdd_coalesced(const float *A, const float *B, float *C, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    __shared__ float A_shared[16];
    __shared__ float B_shared[16];
    A_shared[threadIdx.x] = A[i];
    B_shared[threadIdx.x] = B[i];
    __syncthreads();
    if (i < numElements) {
        C[i] = A_shared[threadIdx.x] + B_shared[threadIdx.x];
    }
}

int main(void) {
    int numElements = 50000;
    size_t size = numElements * sizeof(float);
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    // Initialize the input data
    for (int i = 0; i < numElements; ++i) {
        h_A[i] = i;
        h_B[i] = i;
    }

    // Allocate vectors in device memory
    float *d_A;
    hipMalloc(&d_A, size);
    float *d_B;
    hipMalloc(&d_B, size);
    float *d_C;
    hipMalloc(&d_C, size);

    // Copy vectors from host memory to device memory
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Invoke kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);

    // Copy result from device memory to host memory
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Verify result
    for (int i = 0; i < numElements; ++i) {
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5) {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }

    printf("Test PASSED\n");

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}