#include <hip/hip_runtime.h>

// CUDA kernel for element-wise addition of two matrices
__global__ void matrixAdd(const float *A, const float *B, float *C, int numRows, int numCols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < numRows && col < numCols) {
        int idx = row * numCols + col;
        C[idx] = A[idx] + B[idx];
    }
}   


// CUDA kernel for element-wise addition of two matrices with memory coalescing
__global__ void matrixAdd_coalesced(const float *A, const float *B, float *C, int numRows, int numCols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ int A_shared[16][16];
    __shared__ int B_shared[16][16];
    A_shared[threadIdx.y][threadIdx.x] = A[row * numCols + col];
    B_shared[threadIdx.y][threadIdx.x] = B[row * numCols + col];
    __syncthreads();
    if (row < numRows && col < numCols) {
        int idx = row * numCols + col;
        C[idx] = A_shared[threadIdx.y][threadIdx.x] + B_shared[threadIdx.y][threadIdx.x];
    }
}

 