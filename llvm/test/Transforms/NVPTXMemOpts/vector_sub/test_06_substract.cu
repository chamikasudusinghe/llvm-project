#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>


// CUDA kernel for vector subtraction
__global__ void vectorSubtract(const float *A, const float *B, float *C, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements) {
        C[i] = A[i] - B[i];
    }
}

int main(void) {
    int numElements = 50000;
    size_t size = numElements * sizeof(float);
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    // Initialize the input data
    for (int i = 0; i < numElements; ++i) {
        h_A[i] = i + 10;  // Add 10 to make sure the result is positive
        h_B[i] = i;
    }

    // Allocate vectors in device memory
    float *d_A;
    hipMalloc(&d_A, size);
    float *d_B;
    hipMalloc(&d_B, size);
    float *d_C;
    hipMalloc(&d_C, size);

    // Copy vectors from host memory to device memory
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Invoke kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    vectorSubtract<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);

    // Copy result from device memory to host memory
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Verify result
    for (int i = 0; i < numElements; ++i) {
        if (fabs((h_A[i] - h_B[i]) - h_C[i]) > 1e-5) {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }

    printf("Test PASSED\n");

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}