#include <iostream>
#include <hip/hip_runtime.h>

#define N 4096  // Define matrix size (N x N)

// Naive Matrix-Vector Multiplication
__global__ void matrixVectorMultiplyNaiveDouble(const double *A, const double *B, double *C, int width) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < width) {
        double sum = 0;
        for (int i = 0; i < width; i++) {
            sum += A[id * width + i] * B[i];
        }
        C[id] = sum;
    }
}

// Coalesced Matrix-Vector Multiplication
__global__ void matrixVectorMultiplyCoalescedDouble(const double *A, const double *B, double *C, int width) {
    __shared__ double sharedB[16];
    __shared__ double sharedA[16][16 + 1];
    int tidx = threadIdx.x;
    int bidx = blockIdx.x;
    int idx = bidx * blockDim.x + tidx;
    double sum = 0;

    for (int i = 0; i < width; i += 16) {
        if (i + tidx < width) sharedB[tidx] = B[i + tidx];
        
        for (int l = 0; l < 16; l++) {
            if ((idx - tidx + l < width) && (i + tidx < width)) {
                sharedA[l][tidx] = A[(idx - tidx + l) * width + (i + tidx)];
            }
        }
        __syncthreads();
        for (int k = 0; k < 16; k++) {
            if (i + k < width) {
                sum += sharedA[tidx][k] * sharedB[k];
            }
        }
        __syncthreads(); 
    }
    if (idx < width) C[idx] = sum;
}

// Coalesced and Prefetched Matrix-Vector Multiplication
__global__ void matrixVectorMultiplyCoalescedPrefetchedDouble(const double *A, const double *B, double *C, int width) {
    __shared__ double sharedB[16];
    __shared__ double sharedA[16][16 + 1];
    int tidx = threadIdx.x;
    int bidx = blockIdx.x;
    int idx = bidx * blockDim.x + tidx;
    double sum = 0;
    double prefetchedB = 0;

    if (tidx < width) prefetchedB = B[tidx];

    for (int i = 0; i < width; i += 16) {
        sharedB[tidx] = prefetchedB;
        if (i + 16 + tidx < width) {
            prefetchedB = B[i + 16 + tidx];
        }
        for (int l = 0; l < 16; l++) {
            if ((idx - tidx + l < width) && (i + tidx < width)) {
                sharedA[l][tidx] = A[(idx - tidx + l) * width + (i + tidx)];
            }
        }
        __syncthreads();
        for (int k = 0; k < 16; k++) {
            if (i + k < width) {
                sum += sharedA[tidx][k] * sharedB[k];
            }
        }
        __syncthreads();
    }
    if (idx < width) C[idx] = sum;
}

int main() {
    size_t bytes = N * N * sizeof(double);
    size_t vectorBytes = N * sizeof(double);
    double *h_A = new double[N * N];
    double *h_B = new double[N];
    double *h_C = new double[N];
    double *h_C_coalesced = new double[N];
    double *h_C_prefetched = new double[N];

    // Initialize matrices and vector
    for (int i = 0; i < N * N; i++) {
        h_A[i] = 1.0;
    }
    for (int i = 0; i < N; i++) {
        h_B[i] = 2.0;
    }

    double *d_A, *d_B, *d_C;
    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, vectorBytes);
    hipMalloc(&d_C, vectorBytes);

    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, vectorBytes, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds;

    // Naive matrix-vector multiplication
    hipEventRecord(start);
    matrixVectorMultiplyNaiveDouble<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Naive execution time: " << milliseconds << " ms\n";
    hipMemcpy(h_C, d_C, vectorBytes, hipMemcpyDeviceToHost);

    // Coalesced matrix-vector multiplication
    hipEventRecord(start);
    matrixVectorMultiplyCoalescedDouble<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Coalesced execution time: " << milliseconds << " ms\n";
    hipMemcpy(h_C_coalesced, d_C, vectorBytes, hipMemcpyDeviceToHost);

    // Coalesced and Prefetched matrix-vector multiplication
    hipEventRecord(start);
    matrixVectorMultiplyCoalescedPrefetchedDouble<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Coalesced and Prefetched execution time: " << milliseconds << " ms\n";
    hipMemcpy(h_C_prefetched, d_C, vectorBytes, hipMemcpyDeviceToHost);

    // Verify
    bool correct = true;
    for (int i = 0; i < N; i++) {
        if (h_C[i] != 2 * N || h_C_coalesced[i] != 2 * N || h_C_prefetched[i] != 2 * N) {
            std::cerr << "Error: Vector result is incorrect at index " << i << std::endl;
            correct = false;
            break;
        }
    }
    if (correct) {
        std::cout << "All results are correct." << std::endl;
    }

    // Cleanup
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    delete[] h_C_coalesced;
    delete[] h_C_prefetched;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}