#include <hip/hip_runtime.h>
#include <iostream>

#include <cstdint> // Needed for int64_t

__global__ void naive_reduction(double *input, double *output, int row, int cols) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < cols) {
        double sum = 0.0f;
        for (int i = 0; i < row; i++) {
            sum += input[i * cols + idx];
        }
        output[idx] = sum;
    }
}

__global__ void coalesced(double *input, double *output, int row, int cols) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < cols) {
        double sum = 0.0f;

        for (int i = 0; i < row; i += 16) {
            // Shared memory for coalesced access
            __shared__ double shared[16];
            shared[threadIdx.x] = input[i * cols + idx];
            __syncthreads();
            for (int k = 0; k < 16; k++) {
                sum += shared[k];
            }
            __syncthreads();
        }
        output[idx] = sum;
    }
}
__global__ void coalesced_prefetched(double *input, double *output, int row, int cols) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < cols) {
        double sum = 0.0f;

        // Shared memory for coalesced access
        double tmp = input[threadIdx.x * cols + idx];
        for (int i = 0; i < row; i += 16) {
            __shared__ double shared[16];
            shared[threadIdx.x] = tmp;
            __syncthreads();

            if (i + 16 < row) {
                tmp = input[(i + 16) * cols + idx];
            }

            for (int k = 0; k < 16; k++) {
                sum += shared[k];
            }
            __syncthreads();
        }
        output[idx] = sum;
    }
}



int main() {
    int N = 262144; // Number of rows
    int M = 4096; // Number of columns

    size_t bytes_input = N * M * sizeof(double);
    size_t bytes_output = M * sizeof(double);

    double *h_input = new double[N * M];
    double *h_output = new double[M];
    double *h_output_coalesced = new double[M];
    double *h_output_coalesced_prefetched = new double[M];
    
    // Initialize the input matrix
    for (int i = 0; i < N * M; i++) {
        h_input[i] = 1.0f; // Example initialization
    }

    double *d_input, *d_output;
    hipMalloc(&d_input, bytes_input);
    hipMalloc(&d_output, bytes_output);

    hipMemcpy(d_input, h_input, bytes_input, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(256);
    dim3 numBlocks((M + threadsPerBlock.x - 1) / threadsPerBlock.x);

    // Execute the kernel
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds;

    hipEventRecord(start);
    naive_reduction<<<numBlocks, threadsPerBlock>>>(d_input, d_output, N, M);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Naive execution time: " << milliseconds << " ms\n";
    hipMemcpy(h_output, d_output, bytes_output, hipMemcpyDeviceToHost);

    hipEventRecord(start);
    coalesced<<<numBlocks, threadsPerBlock>>>(d_input, d_output, N, M);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Coalesced execution time: " << milliseconds << " ms\n";
    hipMemcpy(h_output_coalesced, d_output, bytes_output, hipMemcpyDeviceToHost);

    hipEventRecord(start);
    coalesced_prefetched<<<numBlocks, threadsPerBlock>>>(d_input, d_output, N, M);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Coalesced_prefetch execution time: " << milliseconds << " ms\n";
    hipMemcpy(h_output_coalesced_prefetched, d_output, bytes_output, hipMemcpyDeviceToHost);


    // Output the results
    for (int i = 0; i < M; i++) {
        if (h_output[i] != N || h_output_coalesced[i] != N || h_output_coalesced_prefetched[i] != N) {
            std::cout << "Error at index " << i << std::endl;
            std::cout << "Naive: " << h_output[i] << std::endl;
            std::cout << "Coalesced: " << h_output_coalesced[i] << std::endl;
            std::cout << "Coalesced Prefetched: " << h_output_coalesced_prefetched[i] << std::endl;
        }
    }

    // Cleanup
    hipFree(d_input);
    hipFree(d_output);
    delete[] h_input;
    delete[] h_output;
    delete[] h_output_coalesced;
    delete[] h_output_coalesced_prefetched;

    return 0;
}