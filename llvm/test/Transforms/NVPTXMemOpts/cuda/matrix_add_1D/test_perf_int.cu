#include <hip/hip_runtime.h>
#include <iostream>

#include <cstdint> // Needed for int64_t

// CUDA kernel for element-wise addition of two matrices
__global__ void matrixAdd(const int64_t *A, const int64_t *B, int64_t *C, int numRows, int numCols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < numRows && col < numCols) {
        int idx = row * numCols + col;
        C[idx] = A[idx] + B[idx];
    }
}   


// CUDA kernel for element-wise addition of two matrices with memory coalescing
__global__ void matrixAdd_coalesced(const int64_t *A, const int64_t *B, int64_t *C, int numRows, int numCols) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ int A_shared[16][16];
    __shared__ int B_shared[16][16];
    A_shared[threadIdx.y][threadIdx.x] = A[row * numCols + col];
    B_shared[threadIdx.y][threadIdx.x] = B[row * numCols + col];
    __syncthreads();
    if (row < numRows && col < numCols) {
        int idx = row * numCols + col;
        C[idx] = A_shared[threadIdx.y][threadIdx.x] + B_shared[threadIdx.y][threadIdx.x];
    }
}

 

int main() {
    size_t N = 4096;
    size_t bytes = N * N * sizeof(int64_t);
    int64_t *h_A = new int64_t[N * N];
    int64_t *h_B = new int64_t[N * N];
    int64_t *h_C = new int64_t[N * N]; 
    int64_t *h_C_coalesced = new int64_t[N * N];  

    // Initialize matrices
    for (int i = 0; i < N * N; i++) {
        h_A[i] = 1; 
        h_B[i] = 2;
    }

    int64_t *d_A, *d_B, *d_C;
    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, bytes);
    hipMalloc(&d_C, bytes);

    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks(N / threadsPerBlock.x, N / threadsPerBlock.y);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds;

    // Naive matrix multiplication
    hipEventRecord(start);
    matrixAdd<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, N, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Naive execution time: " << milliseconds << " ms\n";
    hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost);

    // Coalesced matrix multiplication
    hipEventRecord(start);
    matrixAdd_coalesced<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, N, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Coalesced execution time: " << milliseconds << " ms\n";
    hipMemcpy(h_C_coalesced, d_C, bytes, hipMemcpyDeviceToHost);

    // Verify correctness
    bool correct = true;
    for (int i = 0; i < N * N; i++) {
        if (h_C[i] != 3 || h_C_coalesced[i] != 3) {
            std::cerr << "Error: Matrix result is incorrect at index " << i << std::endl;
            correct = false;
            break;
        }
    }
    if (correct) {
        std::cout << "All results are correct." << std::endl;
    }

    // Cleanup
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    delete[] h_C_coalesced;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}