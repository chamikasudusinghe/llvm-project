#include <iostream>
#include <hip/hip_runtime.h>

#define N 4096  // Define matrix size (N x N)

// Naive Matrix-Vector Multiplication
__global__ void matrixVectorMultiplyNaive(const float *A, const float *B, float *C, int width) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if (id < width) {
        float sum = 0;
        for (int i = 0; i < width; i++) {
            sum += A[id * width + i] * B[i];
        }
        C[id] = sum;
    }
}

// Coalesced Matrix-Vector Multiplication
__global__ void matrixVectorMultiplyCoalesced(const float *A, const float *B, float *C, int width) {
    // Shared memory for the vector
    __shared__ float sharedB[16];      
    // Shared memory for the matrix
    __shared__ float sharedA[16][16 + 1];  
    int tidx = threadIdx.x;
    int bidx = blockIdx.x;
    int idx = bidx * blockDim.x + tidx;
    float sum = 0;

    // Loop over the A and B matrices
    for (int i = 0; i < width; i += 16) {
        // Load the vector B into shared memory
        if (i + tidx < width) sharedB[tidx] = B[i + tidx];
        
        // Load the matrix A into shared memory
        for (int l = 0; l < 16; l++) {
            if ((idx - tidx + l < width) && (i + tidx < width)) {
                sharedA[l][tidx] = A[(idx - tidx + l) * width + (i + tidx)];
            }
        }
        __syncthreads();
        for (int k = 0; k < 16; k++) {
            if (i + k < width) {
                sum += sharedA[tidx][k] * sharedB[k];
            }
        }
        __syncthreads(); 
    }
    if (idx < width) C[idx] = sum;
}

// Coalesced and Prefetched Matrix-Vector Multiplication
__global__ void matrixVectorMultiplyCoalescedPrefetched(const float *A, const float *B, float *C, int width) {
    __shared__ float sharedB[16];      
    __shared__ float sharedA[16][16 + 1];
    int tidx = threadIdx.x;
    int bidx = blockIdx.x;
    int idx = bidx * blockDim.x + tidx;

    float sum = 0;
    float prefetchedB = 0;
    // Initial prefetch for the first tile
    if (tidx < width) prefetchedB = B[tidx];

    // Loop over the A and B matrices
    for (int i = 0; i < width; i += 16) {
        // Load the prefetched value into shared memory
        sharedB[tidx] = prefetchedB;
        // Prefetch the next vector
        if (i + 16 + tidx < width) {
            prefetchedB = B[i + 16 + tidx];
        }
        // Load the matrix A into shared memory
        for (int l = 0; l < 16; l++) {
            if ((idx - tidx + l < width) && (i + tidx < width)) {
                sharedA[l][tidx] = A[(idx - tidx + l) * width + (i + tidx)];
            }
        }
        __syncthreads();
        // Perform the multiplication using the shared memory
        for (int k = 0; k < 16; k++) {
            if (i + k < width) {
                sum += sharedA[tidx][k] * sharedB[k];
            }
        }
        __syncthreads();
    }
    if (idx < width) C[idx] = sum;
}

int main() {
    size_t bytes = N * N * sizeof(float);
    size_t vectorBytes = N * sizeof(float);
    float *h_A = new float[N * N];
    float *h_B = new float[N];
    float *h_C = new float[N]; 
    float *h_C_coalesced = new float[N];
    float *h_C_prefetched = new float[N];

    // Initialize matrices and vector
    for (int i = 0; i < N * N; i++) {
        h_A[i] = 1.0f;
    }
    for (int i = 0; i < N; i++) {
        h_B[i] = 2.0f;
    }

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, bytes);
    hipMalloc(&d_B, vectorBytes);
    hipMalloc(&d_C, vectorBytes);

    hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, vectorBytes, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16);
    dim3 numBlocks((N + threadsPerBlock.x - 1) / threadsPerBlock.x);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds;

    // Naive matrix-vector multiplication
    hipEventRecord(start);
    matrixVectorMultiplyNaive<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Naive execution time: " << milliseconds << " ms\n";
    hipMemcpy(h_C, d_C, vectorBytes, hipMemcpyDeviceToHost);

    // Coalesced matrix-vector multiplication
    hipEventRecord(start);
    matrixVectorMultiplyCoalesced<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Coalesced execution time: " << milliseconds << " ms\n";
    hipMemcpy(h_C_coalesced, d_C, vectorBytes, hipMemcpyDeviceToHost);

    // Coalesced and Prefetched matrix-vector multiplication
    hipEventRecord(start);
    matrixVectorMultiplyCoalescedPrefetched<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Coalesced and Prefetched execution time: " << milliseconds << " ms\n";
    hipMemcpy(h_C_prefetched, d_C, vectorBytes, hipMemcpyDeviceToHost);

    // Verify
    bool correct = true;
    for (int i = 0; i < N; i++) {
        if (h_C[i] != 2 * N || h_C_coalesced[i] != 2 * N || h_C_prefetched[i] != 2 * N) {
            std::cerr << "Error: Vector result is incorrect at index " << i << std::endl;
            correct = false;
            break;
        }
    }
    if (correct) {
        std::cout << "All results are correct." << std::endl;
    }

    // Cleanup
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    delete[] h_C_coalesced;
    delete[] h_C_prefetched;

    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}