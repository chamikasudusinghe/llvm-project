#include "hip/hip_runtime.h"
#include <stdio.h>
#include <tb_size_marker.cpp>

// CUDA kernel for vector addition
__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements) {
    __tb_size_marker_1D(256);
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements) {
        C[i] = A[i] + B[i];
    }
}

// Cuda kernel for vector addition with memory coalescing
__global__ void vectorAdd_coalesced(const float *A, const float *B, float *C, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    __shared__ float A_shared[16];
    __shared__ float B_shared[16];
    A_shared[threadIdx.x] = A[i];
    B_shared[threadIdx.x] = B[i];
    __syncthreads();
    if (i < numElements) {
        C[i] = A_shared[threadIdx.x] + B_shared[threadIdx.x];
    }
}
